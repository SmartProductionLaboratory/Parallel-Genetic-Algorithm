#include "hip/hip_runtime.h"
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <include/job_base.h>
#include <vector>
#include <gtest/gtest.h>
#include <iostream>
#include <fstream>

#include <tests/include/test_job_base.h>

#define amount 100000

using namespace std;


class TestJobBaseDevice : public testing::Test{
protected:
	// job_t * jb;
	job_t ** jb_host;
	job_t ** device_jb_addresses;
	job_t ** jb_device;
	unsigned int * result_device;
	unsigned int * result_host;
	double arrayOfMsGene[amount];
	double * array_of_ms_gene_device;
	unsigned int arrayOfSizePt[amount], arrayOfMcNum[amount];
	void SetUp() override;
	void TearDown() override;
	void copyArrayOfJobBase(job_t **, job_t **);
	void setMsGeneData();
};

void TestJobBaseDevice::SetUp() {
	// initialize jb_host_*
	size_t sizeof_array_of_pointer = sizeof(job_t*) * amount;
	size_t sizeof_array_of_result = sizeof(unsigned int) * amount;

	// host memory allocation
	jb_host = (job_t **)malloc(sizeof_array_of_pointer);
	device_jb_addresses = (job_t **)malloc(sizeof_array_of_pointer);
	result_host = (unsigned int *)malloc(sizeof_array_of_result);

	// device memory allocation
	hipMalloc((void **)&jb_device, sizeof_array_of_pointer);
	hipMalloc((void **)&result_device, sizeof_array_of_result);

	setMsGeneData();
	// initializae host array
	for(unsigned int i = 0 ;i < amount; ++i){
		jb_host[i] = newJob(arrayOfSizePt[i]);	
		// jb_host[i] = new JobBaseChild(i);
		// jb_host[i]->setMsGenePointer(&arrayOfMsGene[i]);
		// jb_host[i]->setProcessTime(NULL, arrayOfSizePt[i]);
	}
	//initilize device array
	copyArrayOfJobBase(device_jb_addresses, jb_host);

	// copy content from host to device
	ASSERT_EQ(hipMemcpy(jb_device, device_jb_addresses, sizeof_array_of_pointer, hipMemcpyHostToDevice), hipSuccess);
}

void TestJobBaseDevice::copyArrayOfJobBase(job_t** device_address, job_t** src){
	job_t * device_temp_jb;
	size_t size = sizeof(job_t);
	for(unsigned int i = 0; i < amount; ++i){
		ASSERT_EQ(hipMalloc((void**)&device_temp_jb, size), hipSuccess);
		ASSERT_EQ(hipMemcpy(device_temp_jb, src[i], size, hipMemcpyHostToDevice), hipSuccess);
		device_address[i] = device_temp_jb;
	}
}


void TestJobBaseDevice::TearDown(){
	// delete result_host;

	// for(unsigned int i = 0; i < amount; ++i){
	// 	// free host object
	// 	delete jb_host[i];
	// }
	// 
	// // free array
	// delete [] jb_host;
	// delete [] device_jb_addresses;
	// // free device array
	// ASSERT_EQ(hipFree(jb_device), hipSuccess);
	// ASSERT_EQ(hipFree(result_device), hipSuccess);
}

void TestJobBaseDevice::setMsGeneData(){
    ifstream file;
    file.open("./ms_data.txt", ios::in);
    if (file){
		for(unsigned int i = 0; i < amount; ++i){
			file >> arrayOfMsGene[i] >> arrayOfSizePt[i] >> arrayOfMcNum[i];
		}
        
    }else {
        cout << "Unable to open file\n";
    }
    file.close();
}

__global__ void testMachineSelection(job_t ** jb, unsigned int * result, double * msgene_device, unsigned int numElements){
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < numElements){
		jb[id]->base.init = initJobBase;
		jb[id]->base.init(&jb[id]->base);
		jb[id]->base.setMsGenePointer(&jb[id]->base, &(msgene_device[id]));
		result[id] = jb[id]->base.machineSelection(&jb[id]->base);
	}
}


TEST_F(TestJobBaseDevice, test_machine_selection_host){
	for(int i = 0; i < amount; ++i){
		jb_host[i]->base.setMsGenePointer(&jb_host[i]->base, &arrayOfMsGene[i]);
		ASSERT_EQ(jb_host[i]->base.machineSelection(&jb_host[i]->base), arrayOfMcNum[i]) << "Entry : "<<i<<std::endl;
	}	
}



TEST_F(TestJobBaseDevice, test_machine_selection_device){
	// copy the array content from host to device
	double * msgene_device;
	size_t size_arr = sizeof(double) * amount;
	ASSERT_EQ(hipMalloc((void**)&msgene_device, size_arr), hipSuccess);
	ASSERT_EQ(hipMemcpy(msgene_device, arrayOfMsGene, size_arr, hipMemcpyHostToDevice), hipSuccess);
	// computing
	testMachineSelection<<<256, 1024>>>(jb_device, result_device, msgene_device, amount);
	// copy the array content from device to host
	size_t size = sizeof(unsigned int) * amount;
	ASSERT_EQ(hipMemcpy(result_host, result_device, size, hipMemcpyDeviceToHost), hipSuccess);
	
	// testing
	for(unsigned int i = 0; i < amount; ++i){
		ASSERT_EQ(result_host[i], arrayOfMcNum[i]) << "Entry : " << i << std::endl;
	}
}

