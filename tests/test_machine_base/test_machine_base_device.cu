#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <gtest/gtest.h>
#include <include/linked_list.h>
#include <include/machine_base.h>
#include <include/job_base.h>
#include <include/common.h>

#include "test_machine_base.h"

#define amount 5000

class TestMachineBase : public testing::Test{
public:
	int *values[amount];
	int **result_deivce;
	int **result_device_arr;
	// Machine ** machines;
	Machine ** machines_device_addresses;
	Machine ** machines_device;

	// Job *** jobs;
	Job *** jobs_device;
	Job *** job_device_addresses;
	Job *** job_device_array_addresses;

	unsigned int sizes[amount];
	unsigned int *sizes_device;
	void SetUp();
	// void TearDown();
};

void TestMachineBase::SetUp(){
	Job * job_device_tmp;
	Job * job_tmp;
	Job ** job_device_arr;
	job_device_addresses = (Job ***)malloc(sizeof(Job**)*amount);
	job_device_array_addresses = (Job***)malloc(sizeof(Job**)*amount);


	for(int i = 0; i < amount; ++i){
		sizes[i] = rand() % 100 + 1; // 

		job_device_addresses[i] = (Job **)malloc(sizeof(Job	*)*sizes[i]);
		values[i] = (int*)malloc(sizeof(int)*sizes[i]);
		for(unsigned int j = 0; j < sizes[i]; ++j){
			values[i][j] = rand() % 100;
			job_tmp = newJob(values[i][j]);
			hipMalloc((void**)&job_device_tmp, sizeof(Job));
			hipMemcpy(job_device_tmp, job_tmp, sizeof(Job), hipMemcpyHostToDevice);
			job_device_addresses[i][j] = job_device_tmp;	
		}
		hipMalloc((void**)&job_device_arr, sizeof(Job*)*sizes[i]);
		hipMemcpy(job_device_arr, job_device_addresses[i], sizeof(Job*)*sizes[i], hipMemcpyHostToDevice);
		job_device_array_addresses[i] = job_device_arr;
	}

	hipMalloc((void**)&jobs_device, sizeof(Job**)*amount);
	hipMemcpy(jobs_device, job_device_array_addresses, sizeof(Job**)*amount, hipMemcpyHostToDevice);
	
	hipMalloc((void**)&sizes_device, sizeof(int)*amount);
	hipMemcpy(sizes_device, sizes, sizeof(int)*amount, hipMemcpyHostToDevice);
	

	//***********************************************************************************//
	
	machines_device_addresses = (Machine**)malloc(sizeof(Machine*)*amount);
	Machine *machine_tmp;
	for(int i = 0; i < amount; ++i){
		hipMalloc((void**)&machine_tmp, sizeof(Machine));	
		machines_device_addresses[i] = machine_tmp;
	}
	hipMalloc((void**)&machines_device, sizeof(Machine*)*amount);
	hipMemcpy(machines_device, machines_device_addresses, sizeof(Machine*)*amount, hipMemcpyHostToDevice);

	result_device_arr = (int**)malloc(sizeof(int*)*amount);
	int *result_tmp;
	for(int i = 0; i < amount; ++i){
		hipMalloc((void**)&result_tmp, sizeof(int)*sizes[i]);
		result_device_arr[i] = result_tmp;
	}
	hipMalloc((void**)&result_deivce, sizeof(int*)*amount);
	hipMemcpy(result_deivce, result_device_arr, sizeof(int*)*amount, hipMemcpyHostToDevice);
}

__global__ void initMachinesKernel(Machine ** machines, int am){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < am){
		machines[idx]->base.init = initMachineBase;
		// machines[idx]->base.init(&machines[idx]->base);
		initMachine(machines[idx]);
	}
}

__global__ void initJobsKernel(Job *** jobs, unsigned int *sizes, int am){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < am){
		for(int i = 0; i < sizes[idx];++i){
			initJob(jobs[idx][i]);
		}
	}
}

__global__ void addJobsKernel(Machine ** machines, Job *** jobs, unsigned int *sizes, int **result, int am){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < am){
		for(int i = 0; i < sizes[idx]; ++i){
			machines[idx]->base.addJob(&machines[idx]->base, jobs[idx][i]);		
			// __addJob(&machines[idx]->base, &jobs[idx][i]->ele);
		}
		LinkedListElement *ele;
		ele = machines[idx]->base.root;
		for(unsigned int i = 0; i < sizes[idx] ; ++i){
			result[idx][i] = ele->getValue(ele);
			ele = ele->next;
		}
	}
}

TEST_F(TestMachineBase, test_machine_base_add_job){
	initJobsKernel<<<20, 256>>>(jobs_device, sizes_device, amount);	
	initMachinesKernel<<<20, 256>>>(machines_device, amount);
	addJobsKernel<<<20, 256>>>(machines_device, jobs_device, sizes_device, result_deivce,  amount);

	int *result_tmp;
	int ** arr = (int**)malloc(sizeof(int*)*amount);
	ASSERT_EQ(hipMemcpy(arr, result_deivce, sizeof(int*)*amount, hipMemcpyDeviceToHost), hipSuccess);
	for(int i = 0; i < amount; ++i){
		result_tmp = (int*)malloc(sizeof(int)*sizes[i]);
		ASSERT_EQ(hipMemcpy(result_tmp, arr[i], sizeof(int)*sizes[i], hipMemcpyDeviceToHost), hipSuccess);
		for(int j = 0; j < sizes[i]; ++j){
			ASSERT_EQ(result_tmp[j], values[i][j]);
		}
		free(result_tmp);
	}
}

__global__ void sortJobsKernel(Machine ** machines, Job *** jobs, unsigned int *sizes, int **result, int am){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < am){
		for(int i = 0; i < sizes[idx]; ++i){
			__addJob(&machines[idx]->base, &jobs[idx][i]->ele);
		}
		

		machines[idx]->base.sortJob(&machines[idx]->base);

		LinkedListElement *ele;
		ele = machines[idx]->base.root;
		for(unsigned int i = 0; i < sizes[idx] ; ++i){
			result[idx][i] = jobGetValue(ele);
			ele = ele->next;
		}
		
	}
}

TEST_F(TestMachineBase, test_machine_base_sort_job){
	initJobsKernel<<<20, 256>>>(jobs_device, sizes_device, amount);	
	initMachinesKernel<<<20, 256>>>(machines_device, amount);
	sortJobsKernel<<<20, 256>>>(machines_device, jobs_device, sizes_device, result_deivce,  amount);
	
	int *result_tmp;
	int ** arr = (int**)malloc(sizeof(int*)*amount);
	ASSERT_EQ(hipMemcpy(arr, result_deivce, sizeof(int*)*amount, hipMemcpyDeviceToHost), hipSuccess);
	for(int i = 0; i < amount; ++i){
		result_tmp = (int*)malloc(sizeof(int)*sizes[i]);
		qsort(values[i], sizes[i], sizeof(int), cmpint);
		ASSERT_EQ(hipMemcpy(result_tmp, arr[i], sizeof(int)*sizes[i], hipMemcpyDeviceToHost), hipSuccess);
		for(int j = 0; j < sizes[i]; ++j){
			ASSERT_EQ(result_tmp[j], values[i][j]);
		}
		free(result_tmp);
	}

}
