#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <include/linked_list.h>
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <regex.h>
#include <texture_types.h>
#include <time.h>
#include <tests/def.h>

#define amount 100000

#include "test_linked_list.h"

class TestLinkedListDevice : public testing::Test{
public:
	int *values[amount];

	int ** result_arr;
	int **result_arr_device;

	int **values_arr;
	int **values_arr_device;

	LinkedListItem ***item_address_on_device;
	LinkedListItem *** items_array_of_array;
	LinkedListItem ***items;

	size_t usage;


	int sizes[amount];
	int *sizes_device;
	void SetUp() override;
	void TearDown() override;

	void advanceSetup();

};

void TestLinkedListDevice::SetUp(){

}



void TestLinkedListDevice::advanceSetup(){
	item_address_on_device = (LinkedListItem ***)malloc(sizeof(LinkedListItem**)*amount);	

	LinkedListItem *item_device;
	int count = 0;
	usage = 0;
	for(int i = 0; i < amount; ++i){
		sizes[i] = rand() % 100 + 50;
		values[i] = (int*)malloc(sizeof(int)*sizes[i]);
		count += sizes[i];
		item_address_on_device[i] = (LinkedListItem **)malloc(sizeof(LinkedListItem*)*sizes[i]);
		for(int j = 0; j < sizes[i]; ++j){
			values[i][j] = rand() % 1024;
			ASSERT_EQ(hipMalloc((void**)&item_device, sizeof(LinkedListItem)), hipSuccess);
			item_address_on_device[i][j] = item_device;
		}
	}
	
	usage += count*sizeof(LinkedListItem);
	
	LinkedListItem ** items_array;
	items_array_of_array = (LinkedListItem ***)malloc(sizeof(LinkedListItem **)*amount);
	for(int i = 0; i < amount; ++i){
		ASSERT_EQ(hipMalloc((void**)&items_array, sizeof(LinkedListItem*)*sizes[i]), hipSuccess);
		ASSERT_EQ(hipMemcpy(items_array, item_address_on_device[i], sizeof(LinkedListItem*)*sizes[i], hipMemcpyHostToDevice), hipSuccess);
		items_array_of_array[i] = items_array;

		usage += sizeof(LinkedListItem*)*sizes[i];
	}

	ASSERT_EQ(hipMalloc((void**)&items, sizeof(LinkedListItem**)*amount), hipSuccess);
	ASSERT_EQ(hipMemcpy(items, items_array_of_array, sizeof(LinkedListItem**)*amount, hipMemcpyHostToDevice), hipSuccess);

	usage += sizeof(LinkedListItem**)*amount;
	PRINTF("Amount of testing elements is %d\n", count);
	PRINTF("Average amount of elements handled by a thread is %.2f\n", count / (double)amount);

	int *result_tmp;
	result_arr = (int**)malloc(sizeof(int*)*amount);
	for(int i = 0; i < amount; ++i){
		ASSERT_EQ(hipMalloc((void**)&result_tmp, sizeof(int)*sizes[i]), hipSuccess);
		usage += sizeof(int)*sizes[i];
		result_arr[i] = result_tmp;
	}
	ASSERT_EQ(hipMalloc((void**)&result_arr_device, sizeof(int*)*amount), hipSuccess);
	ASSERT_EQ(hipMemcpy(result_arr_device, result_arr, sizeof(int*)*amount, hipMemcpyHostToDevice), hipSuccess);
	usage += sizeof(int*)*amount;

	ASSERT_EQ(hipMalloc((void**)&sizes_device, sizeof(int)*amount), hipSuccess);
	ASSERT_EQ(hipMemcpy(sizes_device, sizes, sizeof(int)*amount, hipMemcpyHostToDevice), hipSuccess);
	
	usage += sizeof(int)*amount;

	/*********ALLOCATE values array and copy value********/
	values_arr = (int **)malloc(sizeof(int*)*amount);
	int *values_tmp;
	for(int i = 0; i < amount; ++i){
		ASSERT_EQ(hipMalloc((void**)&values_tmp, sizeof(int)*sizes[i]), hipSuccess);
		ASSERT_EQ(hipMemcpy(values_tmp, values[i], sizeof(int)*sizes[i], hipMemcpyHostToDevice), hipSuccess);
		values_arr[i] = values_tmp;
		usage += sizeof(int)*sizes[i];
	}
	ASSERT_EQ(hipMalloc((void**)&values_arr_device, sizeof(int*)*amount), hipSuccess);
	ASSERT_EQ(hipMemcpy(values_arr_device, values_arr, sizeof(int*)*amount, hipMemcpyHostToDevice), hipSuccess);
	usage += sizeof(int*)*amount;

	PRINTF("Device Memory usage : %lu bytes\n", usage);
}

void TestLinkedListDevice::TearDown(){
	// free item
	for(int i = 0; i < amount; ++i){
		for(int j = 0; j < sizes[i]; ++j)
			hipFree(item_address_on_device[i][j]);
		free(item_address_on_device[i]);
		hipFree(items_array_of_array[i]);
	}
	free(items_array_of_array);
	free(item_address_on_device);
	hipFree(items);

	// free result
	for(int i = 0; i < sizes[i]; ++i){
		hipFree(result_arr[i]);
	}
	hipFree(result_arr_device);
	free(result_arr);

	// free sizes
	hipFree(sizes_device);

	// free values
	for(int i = 0; i < amount; ++i){
		hipFree(values_arr[i]);
	}
	hipFree(values_arr_device);
	free(values_arr);
	
}

__global__ void initLinkedListOps(LinkedListElementOperation *ops){
	ops->setNext = __listEleSetNext;
	ops->setPrev = __listEleSetPrev;
}

__global__ void sortingSetUp(LinkedListItem ***items,  int *sizes, int **values){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < amount){
		// first initial all items;
		// connect to device function
		LinkedListElementOperation ops = LINKED_LIST_OPS();
		for(int i = 0; i < sizes[idx]; ++i){
			items[idx][i]->ele.getValue = linkedListItemGetValue;
			// items[idx][i]->ele.setNext = __listEleSetNext;
			// items[idx][i]->ele.setPrev = __listEleSetPrev;
			items[idx][i]->ele.ptr_derived_object = items[idx][i];
			items[idx][i]->value = values[idx][i];
		}

		for(int i = 0, size = sizes[idx] - 1; i < size; ++i){
			ops.setNext(&items[idx][i]->ele, &items[idx][i+1]->ele);		
			// items[idx][i]->ele.setNext(&items[idx][i]->ele, &items[idx][i + 1]->ele);
		}

	}
}

__global__ void sorting(LinkedListItem ***items, int **values, LinkedListElementOperation *ops, int am){
 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
 	if(idx < am){
		LinkedListElement *iter;
 		iter = linkedListMergeSort(&(items[idx][0]->ele), ops);
		items[idx][0] = (LinkedListItem*)iter->ptr_derived_object;
		iter = &(items[idx][0]->ele);

		for(int i = 0; iter ; ++i){
			values[idx][i] = iter->getValue(iter);
			iter = iter->next;
		}
 	}
}



TEST_F(TestLinkedListDevice, test_sort_linked_list_on_device){

	/**********ALLOCAT result array***********/
	advanceSetup();
	/********INIT OPS**********************************/
	LinkedListElementOperation *ops_device;
	ASSERT_EQ(hipMalloc((void**)&ops_device, sizeof(LinkedListElementOperation)), hipSuccess);
	initLinkedListOps<<<1,1>>>(ops_device);
	

	clock_t t1 = clock();
	sortingSetUp<<<1024, 1024>>>(items, sizes_device, values_arr_device);
	sorting<<<1024, 1024>>>(items, result_arr_device, ops_device, amount);
	hipDeviceSynchronize();
	clock_t t2 = clock();
	PRINTF("Time elapse : %.3fs\n", (t2 - t1) / (double)CLOCKS_PER_SEC);
	
	int *result_tmp;
	ASSERT_EQ(hipMemcpy(result_arr, result_arr_device, sizeof(int*)*amount, hipMemcpyDeviceToHost), hipSuccess);
	for(int i = 0; i < amount; ++i){
		result_tmp = (int*)malloc(sizeof(int)*sizes[i]);
		ASSERT_EQ(hipMemcpy(result_tmp, result_arr[i], sizeof(int)*sizes[i], hipMemcpyDeviceToHost), hipSuccess);
		qsort(values[i], sizes[i], sizeof(int), cmpint);
		for(int j = 0; j < sizes[i]; ++j){
			ASSERT_EQ(values[i][j], result_tmp[j]); 
		}
		free(result_tmp);
	}

}
